#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include "ParticlesManager.cuh"
#include "iostream"
#include <hip/hip_runtime.h>
#include ""
#include <thread>
#include <random>

#ifndef PI
#define PI 3.14159265358979323846
#endif


Config* currentConfig;


// CUDA kernel to update particle positions
__global__ void cudaUpdatePositions(Vertex* position, Vertex* velocity, float* mass, short* particleOpacity, Config* config)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= config->N) return;

	Vertex particlePos = position[i];
	float dx = config->mouseX - particlePos.x, dy = config->mouseY - particlePos.y;
	float actualD = std::sqrt(dx * dx + dy * dy);
	float d = fmaxf(actualD, config->minD);
	d = fminf(d, config->maxD);
	float a;

	if (config->version == 1) {
		a = G_CONSTANT * (config->mouseMass / mass[i]) / (d * d);
		float tot = abs(dx) + abs(dy);
		float rX = dx / tot;
		float rY = dy / tot;
		velocity[i].x += rX * a;
		velocity[i].y += rY * a;
	}
	else {
		a = G_CONSTANT * (config->mouseMass) / (d * d);
		velocity[i].x += dx * a / 100.f;
		velocity[i].y += dy * a / 100.f;
	}

	position[i].x += velocity[i].x;
	position[i].y += velocity[i].y;
	float oPercent = fmaxf(abs(a) / 5.f, 0.2);
	particleOpacity[i] = fminf(255, oPercent * 255);

	if (config->attract) {
		float dx = config->mouseX - position[i].x, dy = config->mouseY - position[i].y;
		float tot = abs(dx) + abs(dy);
		position[i].x += dx * config->attractVel / 100;
		position[i].y += dy * config->attractVel / 100;
	}
}


void updateParticles(int start, int end, sf::VertexArray& particles, Vertex* cudaPositions, short* particleOpacity, sf::Color color) {
	for (int i = start; i < end; i++) {
		particles[i].position.x = cudaPositions[i].x;
		particles[i].position.y = cudaPositions[i].y;
		sf::Color pre = particles[i].color;
		particles[i].color = sf::Color(color.r, color.g, color.b, particleOpacity[i]);
	}
}


short randRange(int low, int high) {
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> dist(low, high);
	return dist(gen);
}


sf::Color getNextColor(float t) {
	int r = 220 + 35 * std::sin(PI * t / 2);
	int g = 60 + 50 * std::sin(PI * t / 3 + 2.0);
	int b = 5 + 5 * std::sin(PI * t / 4 + 4.0);
	return sf::Color(r, g, b);
}


ParticlesManager::ParticlesManager(sf::RenderWindow& mWindow, ConfigManager* confManager) {

	cm = confManager;
	attractor.x = 100.f;
	attractor.y = 100.f;
	paused = false;
	sf::Vector2f center = sf::Vector2f(mWindow.getSize().x / 2.0f, mWindow.getSize().y / 2.0f);

	hipMallocManaged(&cudaPositions, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&cudaVelocity, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&mass, NUM_PARTICLES * sizeof(float));
	hipMallocManaged(&currentConfig, sizeof(Config));
	hipMallocManaged(&particleOpacity, NUM_PARTICLES * sizeof(short));

	for (int i = 0; i < NUM_PARTICLES; i++) {
		mass[i] = (std::max((float)MIN_PARTICLE_MASS, ((float)rand() / RAND_MAX) * MAX_PARTICLE_MASS));

		// Spawn randomly in a circular fashion
		float radius = 500 * ((float)rand() / RAND_MAX);
		double theta = ((double)rand() / RAND_MAX) * 2 * PI;
		float x = center.x + radius * cos(theta);
		float y = center.y + radius * sin(theta);

		particles[i].position = sf::Vector2f(x, y);
		// Initial color of particles
		particles[i].color = sf::Color::White;

		cudaPositions[i] = Vertex{ x, y };
		cudaVelocity[i] = Vertex{ 0.f, 0.f };
		particleOpacity[i] = 255;
	}

	t = 0.f;
}


ParticlesManager::~ParticlesManager() {
	hipFree(cudaPositions);
	hipFree(cudaVelocity);
	hipFree(mass);
	hipFree(currentConfig);
	hipFree(particleOpacity);
}


void ParticlesManager::updatePositions(sf::RenderWindow& mWindow, sf::Vector2i& mousePos, bool shouldAttract) {
	if (paused) return;
	// Update config to reflect current condition
	currentConfig->mouseMass = cm->mouseMass;
	currentConfig->mouseX = mousePos.x;
	currentConfig->mouseY = mousePos.y;
	currentConfig->minD = cm->minD;
	currentConfig->maxD = cm->maxD;
	currentConfig->N = NUM_PARTICLES;
	currentConfig->attract = shouldAttract;
	currentConfig->attractVel = cm->attractVel;
	currentConfig->version = cm->version;
	// Run on multiple blocks
	cudaUpdatePositions << <NUM_BLOCKS, BLOCK_SIZE >> > (cudaPositions, cudaVelocity, mass, particleOpacity, currentConfig);
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}


void ParticlesManager::drawParticles(sf::RenderWindow& mWindow) {
	if (paused) {
		mWindow.draw(particles);
		return;
	}
	// Update SFML particles 
	int particlesPerThread = (NUM_PARTICLES / CPU_THREADS) + 1;
	std::vector<std::thread> threads;
	sf::Color newColor = getNextColor(t);

	for (int i = 0; i < CPU_THREADS; i++) {
		int start = i * particlesPerThread;
		int end = std::min(NUM_PARTICLES, start + particlesPerThread);
		threads.emplace_back(updateParticles, start, end, std::ref(particles), cudaPositions, particleOpacity, newColor);
	}
	for (auto& th : threads) {
		th.join();
	}

	t += T_INCREMENT;
	mWindow.draw(particles);
}


void ParticlesManager::togglePaused() {
	paused = !paused;
}


bool ParticlesManager::isPaused() {
	return paused;
}



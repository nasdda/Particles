#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include "ParticlesManager.cuh"
#include "iostream"
#include <hip/hip_runtime.h>
#include ""
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


Config* currentConfig;

ParticlesManager::ParticlesManager(sf::RenderWindow& mWindow, ConfigManager* confManager) {

	cm = confManager;
	attractor.x = 100.f;
	attractor.y = 100.f;
	paused = false;
	sf::Vector2f center = sf::Vector2f(mWindow.getSize().x / 2.0f, mWindow.getSize().y / 2.0f);

	hipMallocManaged(&cudaPositions, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&cudaVelocity, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&mass, NUM_PARTICLES * sizeof(float));
	hipMallocManaged(&currentConfig, sizeof(Config));

	for (int i = 0; i < NUM_PARTICLES; i++) {
		mass[i] = (std::max((float)MIN_PARTICLE_MASS, ((float)rand() / RAND_MAX) * MAX_PARTICLE_MASS));

		// Spawn randomly in a circular fashion
		float radius = 500 * ((float)rand() / RAND_MAX);
		double theta = ((double)rand() / RAND_MAX) * 2 * M_PI;
		float x = center.x + radius * cos(theta);
		float y = center.y + radius * sin(theta);

		particles[i].position = sf::Vector2f(x, y);
		particles[i].color = sf::Color::Black;

		cudaPositions[i] = Vertex{ x, y };
		cudaVelocity[i] = Vertex{ 0.f, 0.f };
	}
}

ParticlesManager::~ParticlesManager() {
	hipFree(cudaPositions);
	hipFree(cudaVelocity);
	hipFree(mass);
	hipFree(currentConfig);
}


// CUDA kernel to update particle positions
__global__ void cudaUpdatePositions(Vertex* position, Vertex* velocity, float* mass, Config *config)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= config->N) return;

	Vertex particlePos = position[i];
	float dx = config->mouseX - particlePos.x, dy = config->mouseY - particlePos.y;
	float d = std::sqrt(dx * dx + dy * dy);
	d = fmaxf(d, config->minD);
	d = fminf(d, config->maxD);
	float a = G * (config->mouseMass / mass[i]) / (d * d);

	float tot = abs(dx) + abs(dy);
	float rX = 0;
	float rY = 0;
	rX = dx / tot;
	rY = dy / tot;

	velocity[i].x += rX * a;
	velocity[i].y += rY * a;
	position[i].x += velocity[i].x;
	position[i].y += velocity[i].y;


	if (config->attract) {
		float dx = config->mouseX - position[i].x, dy = config->mouseY - position[i].y;
		float tot = abs(dx) + abs(dy);
		position[i].x += dx * config->attractVel / 100;
		position[i].y += dy * config->attractVel / 100;
	}
	
}

void ParticlesManager::updatePositions(sf::RenderWindow& mWindow, sf::Vector2i& mousePos, bool shouldAttract) {
	if (paused) return;
	// Update config to reflect current condition
	currentConfig->mouseMass = cm->mouseMass;
	currentConfig->mouseX = mousePos.x;
	currentConfig->mouseY = mousePos.y;
	currentConfig->minD = cm->minD;
	currentConfig->maxD = cm->maxD;
	currentConfig->N = NUM_PARTICLES;
	currentConfig->attract = shouldAttract;
	currentConfig->attractVel = cm->attractVel;
	// Launch kernel on 1M elements on the GPU
	int blockSize = 256;
	int numBlocks = (NUM_PARTICLES + blockSize - 1) / blockSize;
	cudaUpdatePositions<<<numBlocks, blockSize>>>(cudaPositions, cudaVelocity, mass, currentConfig);
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}


void ParticlesManager::drawParticles(sf::RenderWindow& mWindow) {
	for (int i = 0; i < NUM_PARTICLES; i++) {
		particles[i].position = sf::Vector2f(cudaPositions[i].x, cudaPositions[i].y);
	}
	mWindow.draw(particles);
}


void ParticlesManager::togglePaused() {
	paused = !paused;
}

bool ParticlesManager::isPaused() {
	return paused;
}
#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include "ParticlesManager.cuh"
#include "iostream"
#include <hip/hip_runtime.h>
#include ""
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


Config* currentConfig;

// CUDA kernel to update particle positions
__global__ void cudaUpdatePositions(Vertex* position, Vertex* velocity, float* mass, short* particleColors, Config* config)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= config->N) return;

	Vertex particlePos = position[i];
	float dx = config->mouseX - particlePos.x, dy = config->mouseY - particlePos.y;
	float actualD = std::sqrt(dx * dx + dy * dy);
	float d = fmaxf(actualD, config->minD);
	d = fminf(d, config->maxD);
	float a = G_CONSTANT * (config->mouseMass / mass[i]) / (d * d);

	float tot = abs(dx) + abs(dy);
	float rX = 0;
	float rY = 0;
	rX = dx / tot;
	rY = dy / tot;

	velocity[i].x += rX * a;
	velocity[i].y += rY * a;
	position[i].x += velocity[i].x;
	position[i].y += velocity[i].y;
	float oPercent = fmaxf(abs(a) / 5.f, 0.15);
	particleColors[i] = fminf(255, oPercent * 255);

	if (config->attract) {
		float dx = config->mouseX - position[i].x, dy = config->mouseY - position[i].y;
		float tot = abs(dx) + abs(dy);
		position[i].x += dx * config->attractVel / 100;
		position[i].y += dy * config->attractVel / 100;
	}
}


ParticlesManager::ParticlesManager(sf::RenderWindow& mWindow, ConfigManager* confManager) {

	cm = confManager;
	attractor.x = 100.f;
	attractor.y = 100.f;
	paused = false;
	sf::Vector2f center = sf::Vector2f(mWindow.getSize().x / 2.0f, mWindow.getSize().y / 2.0f);

	hipMallocManaged(&cudaPositions, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&cudaVelocity, NUM_PARTICLES * sizeof(Vertex));
	hipMallocManaged(&mass, NUM_PARTICLES * sizeof(float));
	hipMallocManaged(&currentConfig, sizeof(Config));
	hipMallocManaged(&particleColors, NUM_PARTICLES * sizeof(short));

	for (int i = 0; i < NUM_PARTICLES; i++) {
		mass[i] = (std::max((float)MIN_PARTICLE_MASS, ((float)rand() / RAND_MAX) * MAX_PARTICLE_MASS));

		// Spawn randomly in a circular fashion
		float radius = 500 * ((float)rand() / RAND_MAX);
		double theta = ((double)rand() / RAND_MAX) * 2 * M_PI;
		float x = center.x + radius * cos(theta);
		float y = center.y + radius * sin(theta);

		particles[i].position = sf::Vector2f(x, y);
		particles[i].color = sf::Color::Black;

		cudaPositions[i] = Vertex{ x, y };
		cudaVelocity[i] = Vertex{ 0.f, 0.f };
		particleColors[i] = 255;
	}
}

ParticlesManager::~ParticlesManager() {
	hipFree(cudaPositions);
	hipFree(cudaVelocity);
	hipFree(mass);
	hipFree(currentConfig);
	hipFree(particleColors);
}

void ParticlesManager::updatePositions(sf::RenderWindow& mWindow, sf::Vector2i& mousePos, bool shouldAttract) {
	if (paused) return;
	// Update config to reflect current condition
	currentConfig->mouseMass = cm->mouseMass;
	currentConfig->mouseX = mousePos.x;
	currentConfig->mouseY = mousePos.y;
	currentConfig->minD = cm->minD;
	currentConfig->maxD = cm->maxD;
	currentConfig->N = NUM_PARTICLES;
	currentConfig->attract = shouldAttract;
	currentConfig->attractVel = cm->attractVel;
	// Run on multiple blocks
	int blockSize = 256;
	int numBlocks = (NUM_PARTICLES + blockSize - 1) / blockSize;
	cudaUpdatePositions<<<numBlocks, blockSize>>>(cudaPositions, cudaVelocity, mass, particleColors, currentConfig);
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}


void ParticlesManager::drawParticles(sf::RenderWindow& mWindow) {
	for (int i = 0; i < NUM_PARTICLES; i++) {
		particles[i].position.x = cudaPositions[i].x;
		particles[i].position.y = cudaPositions[i].y;
		particles[i].color = sf::Color(0, 170, 255, particleColors[i]);
	}
	mWindow.draw(particles);
}


void ParticlesManager::togglePaused() {
	paused = !paused;
}

bool ParticlesManager::isPaused() {
	return paused;
}


